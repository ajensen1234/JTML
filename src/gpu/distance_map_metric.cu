#include "hip/hip_runtime.h"
/*GPU Metrics Header*/
#include "gpu/gpu_metrics.cuh"

/*Cuda*/
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

/*Grayscale Colors*/
#include "pixel_grayscale_colors.h"

/*Launch Parameters*/
#include "cuda_launch_parameters.h"

#include "gpu/fast_implant_dilation_metric.cuh"


namespace gpu_cost_function{

    __global__ void DistanceMapMetric_Kernel(
        unsigned char* projected_image,
        unsigned char* distance_map,
        int* distance_map_score,
        int* edge_pixel_count,
        int width, int height,
        int diff_kernel_left_x,
        int diff_kernel_bottom_y,
        int diff_kernel_cropped_width
){
        // Global Thread
        int i = (blockIdx.y + gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

        // Convert thread to subsize within the full image
        // i = (i/diff_kernel_cropped_width) * width + (i%diff_kernel_cropped_width)
        //     + (diff_kernel_bottom_y * width) + diff_kernel_left_x;
        int pixel;
        int distance_map_value;
        if (i<width*height){
            atomicAdd(&distance_map_score[0], 1);

            pixel = projected_image[i];
            distance_map_value = distance_map[i];
            if (pixel == EDGE_PIXEL ||  pixel == WHITE_PIXEL){
                atomicAdd(&edge_pixel_count[0], 1);

            }
        }
    }

    double GPUMetrics::DistanceMapMetric(
        GPUImage* projected_image,
        GPUFrame* distance_map,
        int dilation
){
        /*
        This is the distance map metric with the hopes to "convexify" the
        search space a little bit more.
        Instead of just using the Hamming/Dilation overlap as a metric, which has many local min.
        We employ a distance map and calculate the average distance of each projected pixel
        to the template.
            * */

        int height = projected_image->GetFrameHeight();
        int width = projected_image->GetFrameWidth();
        int* bounding_box = projected_image->GetBoundingBox();

        /*Using previously written kernel to reset the metric*/
        FastImplantDilationMetric_ResetPixelScoreKernel<<<1,1>>>(dev_distance_map_score_);
        FastImplantDilationMetric_ResetPixelScoreKernel<<<1,1>>>(dev_edge_pixels_count_);

		int diff_kernel_left_x = max(bounding_box[0] - dilation, 0);
		int diff_kernel_bottom_y = max(bounding_box[1] - dilation, 0);
		int diff_kernel_right_x = min(bounding_box[2] + dilation, width - 1);
		int diff_kernel_top_y = min(bounding_box[3] + dilation, height - 1);
		int diff_kernel_cropped_width = diff_kernel_right_x - diff_kernel_left_x + 1;
		int diff_kernel_cropped_height = diff_kernel_top_y - diff_kernel_bottom_y + 1;

		//dim_grid_image_processing_ = dim3(
		//	ceil(static_cast<double>(diff_kernel_cropped_width) / sqrt(static_cast<double>(threads_per_block))),
		//	ceil(static_cast<double>(diff_kernel_cropped_height) / sqrt(static_cast<double>(threads_per_block))));

        dim_grid_image_processing_ = dim3(
            ceil(sqrt(static_cast<double>(width*height) / static_cast<double>(threads_per_block))),
            ceil(sqrt(static_cast<double>(height*height) / static_cast<double>(threads_per_block)))
);
        DistanceMapMetric_Kernel <<< dim_grid_image_processing_, threads_per_block>>>(
        projected_image->GetDeviceImagePointer(),
            distance_map->GetDeviceImagePointer(),
            dev_distance_map_score_,
            dev_edge_pixels_count_,
            width, height,
            diff_kernel_left_x,
            diff_kernel_bottom_y,
            diff_kernel_cropped_width);

        hipMemcpy(distance_map_score_, dev_distance_map_score_, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(edge_pixels_count_, dev_edge_pixels_count_, sizeof(int), hipMemcpyDeviceToHost);
        std::cout << *distance_map_score_ << std::endl;
    }
} /*end namespace gpu_cost_function*/

#include "gpu/gpu_heatmaps.cuh"

namespace gpu_cost_function{
    GPUHeatmap::GPUHeatmap(int width, int height,
                           int gpu_device, int num_keypoints,
                           unsigned char* host_heatmaps){
        // Start out assuming initialized incorrectly
        initialized_correctly_ = false;

        /*Cuda Error Status*/
        hipGetLastError();
        hipError_t cudaStatus;

        hipSetDevice(gpu_device);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess){
            heatmap_on_gpu_ = false;
        }
        else {
            width_ = width;
            height_ = height;
            device_ = gpu_device;
            num_keypoints_ = num_keypoints;
            dev_heatmap_ = 0;

            hipMalloc((void**)&dev_heatmap_, width_ * height_* num_keypoints_ * sizeof(unsigned char));

            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess){
                heatmap_on_gpu_ = false;
                hipFree(dev_heatmap_);
            } else {
                heatmap_on_gpu_ = true;
            }

            hipMemcpy(dev_heatmap_, host_heatmaps, width_ * height_ * num_keypoints_ * sizeof(unsigned char),
                       hipMemcpyHostToDevice);
            cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess){
                initialized_correctly_ = false;
                heatmap_on_gpu_ = false;
                hipFree(dev_heatmap_);
                return;
            }
            initialized_correctly_ = true;
            heatmap_on_gpu_ = true;
            return;
        }

    };
    GPUHeatmap::~GPUHeatmap(){
        hipFree(dev_heatmap_);
    };
    unsigned char* GPUHeatmap::GetDeviceHeatmapPointer(){
        return dev_heatmap_;
    };
    int GPUHeatmap::GetFrameWidth(){
        return width_;
    };
    int GPUHeatmap::GetFrameHeight(){
        return height_;
    };
    int GPUHeatmap::GetNumKeypoints(){
        return num_keypoints_;
    };
    bool GPUHeatmap::IsInitializedCorrectly(){
        return initialized_correctly_;
    };
}

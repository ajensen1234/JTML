#include "hip/hip_runtime.h"
/*GPU Metrics Header*/
#include "gpu/gpu_metrics.cuh"

/*Launch Parameters*/
#include "cuda_launch_parameters.h"

/*Grayscale Colors*/
#include "pixel_grayscale_colors.h"

namespace gpu_cost_function {

	/*Constructor and Destructor for GPU Metrics Class*/
	GPUMetrics::GPUMetrics() {

		/*Initialized Correctly?*/
		initialized_correctly_ = true;

		/*Initialize Pinned Memory for Slightly Faster Transfer if Using Mismatched Pixel Count*/
		hipHostAlloc((void**)&pixel_score_, sizeof(int), hipHostMallocDefault);
		if (hipGetLastError() != hipSuccess)
			initialized_correctly_ = false;

		/*Initialize Pinned Memory for IOU Intermediary */
		hipHostAlloc((void**)&intersection_score_, sizeof(int), hipHostMallocDefault);
		if (hipGetLastError() != hipSuccess)
			initialized_correctly_ = false;

		/*Initialize Pinned Memory for IOU Intermediary */
		hipHostAlloc((void**)&union_score_, sizeof(int), hipHostMallocDefault);
		if (hipGetLastError() != hipSuccess)
			initialized_correctly_ = false;

		/*Allocate GPU buffers for pixel score.*/
		hipMalloc((void**)&dev_pixel_score_, sizeof(int));
		if (hipGetLastError() != hipSuccess)
			initialized_correctly_ = false;

		/*Allocate GPU buffers for IOU Intermediary*/
		hipMalloc((void**)&dev_intersection_score_, sizeof(int));
		if (hipGetLastError() != hipSuccess)
			initialized_correctly_ = false;

		/*Allocate GPU buffers for IOU Intermediary*/
		hipMalloc((void**)&dev_union_score_, sizeof(int));
		if (hipGetLastError() != hipSuccess)
			initialized_correctly_ = false;

		/*Allocate GPU buffers for comparison white pixel count.*/
		hipMalloc((void**)&dev_white_pix_count_, sizeof(int));
		if (hipGetLastError() != hipSuccess)
			initialized_correctly_ = false;


		/*Upload (Reset) white pixel count for comparison image from Host to Device.*/
		white_pix_count_ = 0;
		hipMemcpy(dev_white_pix_count_, &white_pix_count_, sizeof(int), hipMemcpyHostToDevice);
		if (hipGetLastError() != hipSuccess)
			initialized_correctly_ = false;


        hipHostAlloc((void**)&distance_map_score_, sizeof(int), hipHostMallocDefault);
		if (hipGetLastError() != hipSuccess)
			initialized_correctly_ = false;
        // Allocate some memory for the dev dm score
        hipMalloc((void**)&dev_distance_map_score_, sizeof(int));
		if (hipGetLastError() != hipSuccess){
			initialized_correctly_ = false;
        }


        // Allocating memory for the edge pixels count (GPU and CPU)
        hipMalloc((void**)&dev_edge_pixels_count_, sizeof(int));
		if (hipGetLastError() != hipSuccess){
			initialized_correctly_ = false;
        }
        hipHostAlloc((void**)&edge_pixels_count_, sizeof(int), hipHostMallocDefault);
		if (hipGetLastError() != hipSuccess){
			initialized_correctly_ = false;
        }




	};

	GPUMetrics::~GPUMetrics() {
		/*Free CUDA*/
		hipFree(dev_pixel_score_);
		hipFree(dev_intersection_score_);
		hipFree(dev_union_score_);
        hipFree(dev_edge_pixels_count_);
        hipFree(dev_distance_map_score_);
        hipFree(dev_curvature_hausdorf_score_);

		/*Free Host*/
		hipHostFree(pixel_score_);
		hipHostFree(intersection_score_);
		hipHostFree(union_score_);
        hipHostFree(distance_map_score_);
        hipHostFree(edge_pixels_count_);
        hipHostFree(curvature_hausdorf_score_);
	};

	/*Reset White Pixel Count*/
	__global__ void ComputeSumWhitePixels__ResetWhitePixelScoreKernel(int* dev_white_pix_count_) {
		dev_white_pix_count_[0] = 0;
	}

	/*CUDA White Pixel Sum Function*/
	__global__ void WhitePixelSum(unsigned char* dev_dilation_comparison_image, int* dev_comparison_white_pix_count,
	                              int width, int height) {
		int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

		if (i < width * height) {
			if (dev_dilation_comparison_image[i] == WHITE_PIXEL)
				atomicAdd(&dev_comparison_white_pix_count[0], 1);
		}
	};

	/*Computes Sum of White Pixels in Image*/
	int GPUMetrics::ComputeSumWhitePixels(GPUImage* image, hipError_t* error) {

		/*Reset Errors*/
		hipGetLastError();

		/*Reset White Pixel Count*/
		ComputeSumWhitePixels__ResetWhitePixelScoreKernel << <1, 1 >> >(dev_white_pix_count_);

		/*Get Sum of White Pixels in Dilation Comparison Image and Total Pixel Sum*/
		auto dim_grid_comparison_white_pix = dim3(
			ceil(sqrt(
				static_cast<double>(image->GetFrameWidth() * image->GetFrameHeight()) / static_cast<double>(256))),
			ceil(sqrt(
				static_cast<double>(image->GetFrameWidth() * image->GetFrameHeight()) / static_cast<double>(256))));
		WhitePixelSum << <dim_grid_comparison_white_pix, 256 >> >(image->GetDeviceImagePointer(), dev_white_pix_count_,
		                                                          image->GetFrameWidth(), image->GetFrameHeight());
		hipMemcpy(&white_pix_count_, dev_white_pix_count_, sizeof(int), hipMemcpyDeviceToHost);
		/*Get Errors*/
		*error = hipGetLastError();
		return white_pix_count_;

	};


	/*Get Initialized Correctly*/
	bool GPUMetrics::IsInitializedCorrectly() {
		return initialized_correctly_;
	}
}
